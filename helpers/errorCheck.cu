
#include <iostream>
#include "errorCheck.h"

void checkError(hipError_t errSync, const char *func, const char *file, const int line)
{
    // Check for last Asynchronous error in CUDA
    hipError_t errAsync = hipGetLastError();
    if (errAsync != hipSuccess)
    {
        std::cout << "CUDA Runtime Async Error at: " << file << ":" << line
                  << std::endl
                  << hipGetErrorString(errAsync) << std::endl;

        std::exit(EXIT_FAILURE);
    }

    // Check for last Synchronous error in CUDA
    if (errSync != hipSuccess)
    {
        std::cout << "CUDA Runtime Sync Error at: " << file << ":" << line << std::endl
                  << hipGetErrorString(errSync) << " " << func << std::endl;

        std::exit(EXIT_FAILURE);
    }

    return;
}