// Include the header files needed
#include <iostream>

#include "../gpuKernels/basicKernels.h"
#include "../gpuKernels/componentKernels.h"
#include "../helpers/errorCheck.h"
#include "../helpers/verifyResults.h"
#include "../helpers/gradients.h"

using real = double;

int main(int argc, char **argv)
{
    // Initialize cuda error object
    hipError_t status;

    // Reset the device before starting
    status = hipDeviceReset();
    checkError(status, "hipDeviceReset", __FILE__, __LINE__);

    // Print results of input data and verification
    int print = 0;

    // Add random material properties - copy to constant memory
    real lambda = 55.27e9;
    real mu = 25.95e9;

    std::cout << "\n---------- Constitutive update ----------" << std::endl
              << "Material properties: " << std::endl
              << "Lambda value:     " << lambda << std::endl
              << "Mu value:         " << mu << "\n"
              << std::endl;

    // Determine the number of elements and quadrature points per element
    int elem = atoi(argv[1]) * 1e6;
    int quads = 1;
    int nEval = elem * quads;

    // Log how many iterations are performed
    std::cout << "\n---------- Launch Configuration ----------" << std::endl
              << "Number of quadrature points:      " << quads << std::endl
              << "Number of elements :              " << elem / 1e6 << " M " << std::endl
              << "Number of constitutive updates:   " << nEval / 1e6 << " M" << std::endl;

    // Allocate workers
    int blockSize, gridSize, activeLastBlock;
    if (nEval > 512)
    {
        blockSize = 512;
        gridSize = nEval / blockSize + 1;
        activeLastBlock = nEval - (gridSize - 1) * blockSize;
        if (activeLastBlock == 0)
        {
            gridSize -= 1;
            activeLastBlock = blockSize;
        }
    }
    else
    {
        blockSize = nEval;
        gridSize = 1;
        activeLastBlock = blockSize;
    }

    // Show a summary of the grid that is used and its allocated threads
    std::cout << "\nNumber of threads per block:      " << blockSize << std::endl
              << "Number of blocks:                 " << gridSize << std::endl
              << "Number of threads:                " << gridSize * blockSize << std::endl
              << "Active threads in last block:     " << activeLastBlock << std::endl;

    // Initialize variables to be stored in the GPU
    real *Fg, *Pg;

    // Allocate Unified Memory – accessible from CPU or GPU
    status = hipMallocManaged(&Fg, nEval * 9 * sizeof(real));
    checkError(status, "hipMallocManaged", __FILE__, __LINE__);

    status = hipMallocManaged(&Pg, nEval * 9 * sizeof(real));
    checkError(status, "hipMallocManaged", __FILE__, __LINE__);

    // Define an an array of random deformation gradients as a perturbation of the identity
    real F_pert = 0.1;
    getStridedGradients(F_pert, nEval, Fg, print);

    // Create cuda timers
    hipEvent_t startComp, stopComp;
    status = hipEventCreate(&startComp);
    checkError(status, "hipEventCreate", __FILE__, __LINE__);
    status = hipEventCreate(&stopComp);
    checkError(status, "hipEventCreate", __FILE__, __LINE__);

    std::cout << "\n---------- Cuda Event Timing ----------" << std::endl;

    // Executing computation
    status = hipEventRecord(startComp);
    checkError(status, "hipEventRecord", __FILE__, __LINE__);
    std::cout << "\nSynchronizing device..." << std::endl;

    // Running the strided unified memory implementation on the GPU
    std::cout << "Running strided unified implementation" << std::endl;
    stridedConstitutive<<<gridSize, blockSize>>>(Fg, lambda, mu, Pg, nEval);

    // Wait for GPU to finish before accessing on host
    status = hipDeviceSynchronize();
    checkError(status, "hipDeviceSynchronize", __FILE__, __LINE__);

    // Check timing of executing computation
    status = hipEventRecord(stopComp);
    checkError(status, "hipEventRecord", __FILE__, __LINE__);
    status = hipEventSynchronize(stopComp);
    checkError(status, "hipEventSynchronize", __FILE__, __LINE__);

    float milliseconds = 0.;
    status = hipEventElapsedTime(&milliseconds, startComp, stopComp);
    checkError(status, "hipEventElapsedTime", __FILE__, __LINE__);

    std::cout << "Finished synchronizing device" << std::endl
              << "\nExecuting computation took: " << milliseconds << " ms" << std::endl;

    // Destroy events
    status = hipEventDestroy(startComp);
    checkError(status, "hipEventDestroy", __FILE__, __LINE__);
    status = hipEventDestroy(stopComp);
    checkError(status, "hipEventDestroy", __FILE__, __LINE__);

    // Verify each result is at the correct index
    std::cout << "\nStarting verification by index..." << std::endl;
    verifyStridedResults(nEval, lambda, mu, Fg, Pg, print);
    std::cout << "Finished verification by index..." << std::endl;

    // Free the allocated memory
    status = hipFree(Fg);
    checkError(status, "hipFree", __FILE__, __LINE__);
    status = hipFree(Pg);
    checkError(status, "hipFree", __FILE__, __LINE__);

    status = hipDeviceReset();
    checkError(status, "hipDeviceReset", __FILE__, __LINE__);

    return 0;
}
