#include "hip/hip_runtime.h"
// Include the header files needed
#include <iostream>

#include "../gpuKernels/symmetryKernels.h"
#include "../gpuKernels/axialKernels.h"
#include "../helpers/errorCheck.h"
#include "../helpers/verifyResults.h"
#include "../helpers/gradients.h"

int main(int argc, char **argv)
{
    // Initialize error status
    hipError_t status;

    // Reset the device before starting
    status = hipDeviceReset();
    checkError(status, "hipDeviceReset", __FILE__, __LINE__);

    // Print results of input data and verification
    int print = 0;

    // Add random material properties - copy to constant memory
    real lambda = 55.27e9;
    real mu = 25.95e9;

    std::cout << "\n---------- Constitutive update ----------" << std::endl
              << "Material properties: " << std::endl
              << "Lambda value:     " << lambda << std::endl
              << "Mu value:         " << mu << "\n"
              << std::endl;

    // Determine the number of elements and quadrature points per element
    int elem = atoi(argv[1]) * 1e6;
    int quads = 1;
    int nEval = elem * quads;
    int nAlloc = nEval * 9;

    // Log how many iterations are performed
    std::cout << "\n---------- Launch Configuration ----------" << std::endl
              << "Number of quadrature points:      " << quads << std::endl
              << "Number of elements :              " << elem / 1e6 << " M " << std::endl
              << "Number of constitutive updates:   " << nEval / 1e6 << " M" << std::endl;

    // Allocate threads for each block
    int nThreadsPerBlock = 512;

    // Set the gridsize (total number of threads) optimal amount varies per problem
    int gridsize = 400000;

    int nBlocks, nThreadsLastBlock, nLoops;
    int nThreadsLastLoop, nBlocksLastLoop;
    if (nEval > nThreadsPerBlock)
    {
        // Retrieve the properties of the GPU
        hipDeviceProp_t prop;
        status = hipGetDeviceProperties(&prop, 0);
        checkError(status, "hipGetDeviceProperties", __FILE__, __LINE__);

        // Determine the instruction size (= amount of threads executed in parallel by all SM's)
        // There are 32 threads per SM
        int instructionSize = prop.multiProcessorCount * 32;

        // Determine the number of blocks
        nBlocks = gridsize / nThreadsPerBlock + 1;

        // Check if the last block is being used
        int nThreadsLastBlock = gridsize - (nBlocks - 1) * nThreadsPerBlock;
        if (nThreadsLastBlock == 0)
        {
            nBlocks -= 1;
            nThreadsLastBlock = nThreadsPerBlock;
        }

        // Determine the amount of loops over the saturation size to complete the problem size
        nLoops = nEval / gridsize + 1;

        // Determine the amount of blocks in the last loop
        nThreadsLastLoop = nEval - (nLoops - 1) * gridsize;

        // Check if the last loop is necessary
        int nThreadsLastLoopBlock;
        if (nThreadsLastLoop == 0)
        {
            nLoops -= 1;
            nBlocksLastLoop = nBlocks;
            nThreadsLastLoopBlock = nThreadsPerBlock;
        }
        else
        {
            // Check if the last block in the last loop is being used
            nBlocksLastLoop = nThreadsLastLoop / nThreadsPerBlock + 1;
            nThreadsLastLoopBlock = nThreadsLastLoop - (nBlocksLastLoop - 1) * nThreadsPerBlock;
            if (nThreadsLastLoopBlock == 0)
            {
                nBlocksLastLoop -= 1;
                nThreadsLastLoopBlock = nThreadsPerBlock;
            }
        }

        std::cout << "\nInstruction size:                 " << instructionSize << std::endl
                  << "Used saturation amount:           " << gridsize << std::endl
                  << "Number of rounds:                 " << nLoops << std::endl
                  << "Number of blocks:                 " << nBlocks << std::endl
                  << "Number of threads:                " << nBlocks * nThreadsPerBlock << std::endl
                  << "Last block:                       " << nThreadsLastBlock << std::endl
                  << "Active blocks in last round:      " << nBlocksLastLoop << std::endl
                  << "Active threads in last round:     " << nBlocksLastLoop * nThreadsPerBlock << std::endl
                  << "Active threads in last block:     " << nThreadsLastLoopBlock << std::endl;
    }
    else
    {
        nThreadsPerBlock = nEval;
        nBlocks = 1;
        nThreadsLastBlock = nThreadsPerBlock;
    }

    // Create cuda timers
    hipEvent_t startComp, stopComp;
    status = hipEventCreate(&startComp);
    checkError(status, "hipEventCreate", __FILE__, __LINE__);
    status = hipEventCreate(&stopComp);
    checkError(status, "hipEventCreate", __FILE__, __LINE__);

    // Initialize variables to be stored in the cpu
    real *F, *P;
    status = hipHostMalloc(&F, nAlloc * sizeof(real));
    checkError(status, "hipHostMalloc", __FILE__, __LINE__);

    status = hipHostMalloc(&P, nAlloc * sizeof(real));
    checkError(status, "hipHostMalloc", __FILE__, __LINE__);

    // Define an an array of random deformation gradients as a perturbation of the identity
    real F_pert = 0.1;
    getCoalescedGradients(F_pert, nEval, F, print);

    // Allocate memory on the GPU
    real *Fg, *Pg;
    status = hipMalloc(&Fg, nAlloc * sizeof(real));
    checkError(status, "hipMalloc", __FILE__, __LINE__);

    status = hipMalloc(&Pg, nAlloc * sizeof(real));
    checkError(status, "hipMalloc", __FILE__, __LINE__);

    // Create cuda streams
    hipStream_t *streams;
    hipMalloc(&streams, 3 * nLoops * sizeof(hipStream_t));
    for (int i = 0; i < (3 * nLoops); i++)
    {
        status = hipStreamCreate(&(streams[i]));
        checkError(status, "hipStreamCreate", __FILE__, __LINE__);
    }

    std::cout << "\n---------- Cuda Event Timing ----------" << std::endl;

    // Executing computation
    status = hipEventRecord(startComp);
    checkError(status, "hipEventRecord", __FILE__, __LINE__);
    std::cout << "\nSynchronizing device..." << std::endl;

    // Compute the diagonal components
    int loopSize = gridsize;
    int loopBlocks = nBlocks;
    for (int i = 0; i < nLoops; i++)
    {
        // Set the size for the last loop
        if (i == (nLoops - 1))
        {
            loopSize = nThreadsLastLoop;
            loopBlocks = nBlocksLastLoop;
        }
        else
        {
            loopSize = gridsize;
            loopBlocks = nBlocks;
        }

        // Send input data for the saturated computation
        status = hipMemcpyAsync(Fg + i * gridsize, F + i * gridsize, loopSize * sizeof(real), hipMemcpyHostToDevice, streams[i * 3]);
        checkError(status, "hipMemcpyAsync", __FILE__, __LINE__);

        status = hipMemcpyAsync(Fg + i * gridsize + 4 * nEval, F + i * gridsize + 4 * nEval, loopSize * sizeof(real), hipMemcpyHostToDevice, streams[i * 3 + 1]);
        checkError(status, "hipMemcpyAsync", __FILE__, __LINE__);

        status = hipMemcpyAsync(Fg + i * gridsize + 8 * nEval, F + i * gridsize + 8 * nEval, loopSize * sizeof(real), hipMemcpyHostToDevice, streams[i * 3 + 2]);
        checkError(status, "hipMemcpyAsync", __FILE__, __LINE__);

        // Synchronize the streams
        hipStreamSynchronize(streams[i * 3]);
        hipStreamSynchronize(streams[i * 3 + 1]);
        hipStreamSynchronize(streams[i * 3 + 2]);

        // Perform the saturated computation of P11
        axial11<<<loopBlocks, nThreadsPerBlock, 0, streams[i * 3]>>>(Fg + i * gridsize, lambda, mu, Pg + i * gridsize, nEval, loopSize);

        // Receive results from the saturated computation
        status = hipMemcpyAsync(P + i * gridsize, Pg + i * gridsize, loopSize * sizeof(real), hipMemcpyDeviceToHost, streams[i * 3]);
        checkError(status, "hipMemcpyAsync", __FILE__, __LINE__);

        // Perform the saturated computation of P22
        axial22<<<loopBlocks, nThreadsPerBlock, 0, streams[i * 3 + 1]>>>(Fg + i * gridsize, lambda, mu, Pg + i * gridsize + 4 * nEval, nEval, loopSize);

        status = hipMemcpyAsync(P + i * gridsize + 4 * nEval, Pg + i * gridsize + 4 * nEval, loopSize * sizeof(real), hipMemcpyDeviceToHost, streams[i * 3 + 1]);
        checkError(status, "hipMemcpyAsync", __FILE__, __LINE__);

        // Perform the saturated computation of P22
        axial33<<<loopBlocks, nThreadsPerBlock, 0, streams[i * 3 + 2]>>>(Fg + i * gridsize, lambda, mu, Pg + i * gridsize + 8 * nEval, nEval, loopSize);

        status = hipMemcpyAsync(P + i * gridsize + 8 * nEval, Pg + i * gridsize + 8 * nEval, loopSize * sizeof(real), hipMemcpyDeviceToHost, streams[i * 3 + 2]);
        checkError(status, "hipMemcpyAsync", __FILE__, __LINE__);
    }

    // Compute the shear components
    int iShear[3] = {1, 2, 5};
    int jShear[3] = {3, 6, 7};
    int iComponent, jComponent;
    for (int shearComp = 0; shearComp < 3; shearComp++)
    {
        // Set the indices of the current shear components
        iComponent = iShear[shearComp];
        jComponent = jShear[shearComp];
        for (int i = 0; i < nLoops; i++)
        {
            // Set the size for the last loop
            if (i == (nLoops - 1))
            {
                loopSize = nThreadsLastLoop;
                loopBlocks = nBlocksLastLoop;
            }
            else
            {
                loopSize = gridsize;
                loopBlocks = nBlocks;
            }

            // Send input data for the saturated computation
            status = hipMemcpyAsync(Fg + i * gridsize + iComponent * nEval, F + i * gridsize + iComponent * nEval, loopSize * sizeof(real), hipMemcpyHostToDevice, streams[i]);
            checkError(status, "hipMemcpyAsync", __FILE__, __LINE__);

            status = hipMemcpyAsync(Fg + i * gridsize + jComponent * nEval, F + i * gridsize + jComponent * nEval, loopSize * sizeof(real), hipMemcpyHostToDevice, streams[i]);
            checkError(status, "hipMemcpyAsync", __FILE__, __LINE__);

            // Perform the saturated computation
            shear<<<loopBlocks, nThreadsPerBlock, 0, streams[i]>>>(Fg + i * gridsize, lambda, mu, Pg + i * gridsize, nEval, iComponent, jComponent, loopSize);

            // Receive results from the saturated computation
            status = hipMemcpyAsync(P + i * gridsize + iComponent * nEval, Pg + i * gridsize + iComponent * nEval, loopSize * sizeof(real), hipMemcpyDeviceToHost, streams[i]);
            checkError(status, "hipMemcpyAsync", __FILE__, __LINE__);
        }
    }

    // Wait for GPU to finish before accessing on host
    status = hipDeviceSynchronize();
    checkError(status, "hipDeviceSynchronize", __FILE__, __LINE__);

    // Check timing of executing computation
    status = hipEventRecord(stopComp);
    checkError(status, "hipEventRecord", __FILE__, __LINE__);
    status = hipEventSynchronize(stopComp);
    checkError(status, "hipEventSynchronize", __FILE__, __LINE__);

    // Compute the elapsed time
    float milliseconds = 0.;
    status = hipEventElapsedTime(&milliseconds, startComp, stopComp);
    checkError(status, "hipEventElapsedTime", __FILE__, __LINE__);

    std::cout << "Finished synchronizing device" << std::endl
              << "\nExecuting computation took: " << milliseconds << " ms" << std::endl;

    std::cout << "\n---------- Cuda Stop Timing ----------" << std::endl;

    // Destroy events
    status = hipEventDestroy(startComp);
    checkError(status, "hipEventDestroy", __FILE__, __LINE__);
    status = hipEventDestroy(stopComp);
    checkError(status, "hipEventDestroy", __FILE__, __LINE__);

    // Destroy the cuda streams
    for (int i = 0; i < (3 * nLoops); i++)
    {
        status = hipStreamDestroy(streams[i]);
        checkError(status, "hipStreamDestroy", __FILE__, __LINE__);
    }

    // Verify each result is at the correct index
    std::cout << "\nStarting verification by index..." << std::endl;
    verifyCoalescedResults(nEval, lambda, mu, F, P, print);
    std::cout << "Finished verification by index..." << std::endl;

    // Free the allocated memory
    status = hipFree(Fg);
    checkError(status, "hipFree", __FILE__, __LINE__);

    status = hipFree(Pg);
    checkError(status, "hipFree", __FILE__, __LINE__);

    status = hipHostFree(F);
    checkError(status, "hipHostFree", __FILE__, __LINE__);

    status = hipHostFree(P);
    checkError(status, "hipHostFree", __FILE__, __LINE__);

    // Reset the device in the end
    status = hipDeviceReset();
    checkError(status, "hipDeviceReset", __FILE__, __LINE__);

    return 0;
}