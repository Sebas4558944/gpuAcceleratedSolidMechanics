// Include the header files needed
#include <iostream>

#include "../gpuKernels/basicKernels.h"
#include "../gpuKernels/componentKernels.h"
#include "../helpers/errorCheck.h"
#include "../helpers/verifyResults.h"
#include "../helpers/gradients.h"

int main(int argc, char **argv)
{
    // Initialize cuda error object
    hipError_t status;

    // Reset the device before starting
    status = hipDeviceReset();
    checkError(status, "hipDeviceReset", __FILE__, __LINE__);

    // Print results of input data and verification
    int print = 0;

    // Add random material properties - copy to constant memory
    real lambda = 55.27e9;
    real mu = 25.95e9;

    std::cout << "\n---------- Constitutive update ----------" << std::endl
              << "Material properties: " << std::endl
              << "Lambda value:     " << lambda << std::endl
              << "Mu value:         " << mu << "\n"
              << std::endl;

    // Determine the number of elements and quadrature points per element
    int elem = atoi(argv[1]) * 1e6;
    int quads = 1;
    int nEval = elem * quads;

    // Log how many iterations are performed
    std::cout << "\n---------- Launch Configuration ----------" << std::endl
              << "Number of quadrature points:      " << quads << std::endl
              << "Number of elements :              " << elem / 1e6 << " M " << std::endl
              << "Number of constitutive updates:   " << nEval / 1e6 << " M" << std::endl;

    // Allocate workers
    int blockSize, gridSize, lastUse;
    if (nEval > 512)
    {
        blockSize = 512;
        gridSize = nEval / blockSize + 1;
        lastUse = nEval - (gridSize - 1) * blockSize;
        if (lastUse == 0)
        {
            gridSize -= 1;
            lastUse = blockSize;
        }
    }
    else
    {
        blockSize = nEval;
        gridSize = 1;
        lastUse = blockSize;
    }

    // Show a summary of the grid that is used and its allocated threads
    std::cout << "\nNumber of threads per block:      " << blockSize << std::endl
              << "Number of blocks:                 " << gridSize << std::endl
              << "Number of threads:                " << gridSize * blockSize << std::endl
              << "Active threads in last block:     " << lastUse << std::endl;

    // Initialize variables to be stored in the GPU
    real *F, *P, *Fg, *Pg;

    // Allocate CPU memory
    status = hipHostMalloc(&F, nEval * 9 * sizeof(real));
    checkError(status, "hipHostMalloc", __FILE__, __LINE__);
    status = hipHostMalloc(&P, nEval * 9 * sizeof(real));
    checkError(status, "hipHostMalloc", __FILE__, __LINE__);

    // Allocate GPU memory
    status = hipMalloc(&Fg, nEval * 9 * sizeof(real));
    checkError(status, "hipMalloc", __FILE__, __LINE__);
    status = hipMalloc(&Pg, nEval * 9 * sizeof(real));
    checkError(status, "hipMalloc", __FILE__, __LINE__);

    // Define an an array of random deformation gradients as a perturbation of the identity
    real F_pert = 0.1;
    getCoalescedGradients(F_pert, nEval, F, print);

    // Create cuda timers
    hipEvent_t startComp, stopComp, startSend, stopSend;
    status = hipEventCreate(&startComp);
    checkError(status, "hipEventCreate", __FILE__, __LINE__);
    status = hipEventCreate(&stopComp);
    checkError(status, "hipEventCreate", __FILE__, __LINE__);
    status = hipEventCreate(&startSend);
    checkError(status, "hipEventCreate", __FILE__, __LINE__);
    status = hipEventCreate(&stopSend);
    checkError(status, "hipEventCreate", __FILE__, __LINE__);

    std::cout << "\n---------- Cuda Event Timing ----------" << std::endl;

    // Sending memory and timing the transaction
    status = hipEventRecord(startSend);
    checkError(status, "hipEventRecord", __FILE__, __LINE__);

    hipMemcpy(Fg, F, nEval * 9 * sizeof(real), hipMemcpyHostToDevice);

    status = hipEventRecord(stopSend);
    checkError(status, "hipEventRecord", __FILE__, __LINE__);
    status = hipEventSynchronize(stopSend);
    checkError(status, "hipEventSynchronize", __FILE__, __LINE__);

    // Check the timing
    float milliseconds = 0.;
    status = hipEventElapsedTime(&milliseconds, startSend, stopSend);
    checkError(status, "hipEventElapsedTime", __FILE__, __LINE__);

    std::cout << "\nSending memory took: " << milliseconds << " ms" << std::endl;

    // Executing computation
    status = hipEventRecord(startComp);
    checkError(status, "hipEventRecord", __FILE__, __LINE__);

    std::cout << "\nSynchronizing device..." << std::endl;

    // Running the coalesced pinned memory implementation on the GPU
    std::cout << "Running coalesced pinned implementation" << std::endl;
    coalescedConstitutive<<<gridSize, blockSize>>>(Fg, lambda, mu, Pg, nEval);

    // Wait for GPU to finish before accessing on host
    status = hipDeviceSynchronize();
    checkError(status, "hipDeviceSynchronize", __FILE__, __LINE__);

    // Check timing of executing computation
    status = hipEventRecord(stopComp);
    checkError(status, "hipEventRecord", __FILE__, __LINE__);
    status = hipEventSynchronize(stopComp);
    checkError(status, "hipEventSynchronize", __FILE__, __LINE__);

    milliseconds = 0.;
    status = hipEventElapsedTime(&milliseconds, startComp, stopComp);
    checkError(status, "hipEventElapsedTime", __FILE__, __LINE__);

    std::cout << "\nExecuting computation took: " << milliseconds << " ms" << std::endl
              << "Finished synchronizing device" << std::endl;

    // Receiving memory and timing the transaction
    status = hipEventRecord(startSend);
    checkError(status, "hipEventRecord", __FILE__, __LINE__);

    hipMemcpy(P, Pg, nEval * 9 * sizeof(real), hipMemcpyDeviceToHost);

    status = hipEventRecord(stopSend);
    checkError(status, "hipEventRecord", __FILE__, __LINE__);
    status = hipEventSynchronize(stopSend);
    checkError(status, "hipEventSynchronize", __FILE__, __LINE__);

    // Check the timing
    milliseconds = 0.;
    status = hipEventElapsedTime(&milliseconds, startSend, stopSend);
    checkError(status, "hipEventElapsedTime", __FILE__, __LINE__);

    std::cout << "\nReceiving memory took: " << milliseconds << " ms" << std::endl
              << "\n---------- Cuda Stop Timing ----------" << std::endl;

    // Destroy events
    status = hipEventDestroy(startComp);
    checkError(status, "hipEventDestroy", __FILE__, __LINE__);
    status = hipEventDestroy(stopComp);
    checkError(status, "hipEventDestroy", __FILE__, __LINE__);
    status = hipEventDestroy(startSend);
    checkError(status, "hipEventDestroy", __FILE__, __LINE__);
    status = hipEventDestroy(stopSend);
    checkError(status, "hipEventDestroy", __FILE__, __LINE__);

    // Verify each result is at the correct index
    std::cout << "\nStarting verification by index..." << std::endl;
    verifyCoalescedResults(nEval, lambda, mu, F, P, print);
    std::cout << "Finished verification by index..." << std::endl;

    // Free the allocated memory
    status = hipHostFree(F);
    checkError(status, "hipHostFree", __FILE__, __LINE__);
    status = hipHostFree(P);
    checkError(status, "hipHostFree", __FILE__, __LINE__);
    status = hipFree(Fg);
    checkError(status, "hipFree", __FILE__, __LINE__);
    status = hipFree(Pg);
    checkError(status, "hipFree", __FILE__, __LINE__);

    status = hipDeviceReset();
    checkError(status, "hipDeviceReset", __FILE__, __LINE__);

    return 0;
}
