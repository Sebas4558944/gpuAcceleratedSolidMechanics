#include "hip/hip_runtime.h"
// Include the header files needed
#include <iostream>

#include "../gpuKernels/componentKernels.h"
#include "../helpers/errorCheck.h"
#include "../helpers/verifyResults.h"
#include "../helpers/gradients.h"

int main(int argc, char **argv)
{
    // Initialize error status
    hipError_t status;

    // Reset the device before starting
    status = hipDeviceReset();
    checkError(status, "hipDeviceReset", __FILE__, __LINE__);

    // Print results of input data and verification
    int print = 0;

    // Determine the type of memory that is used for the streams (0 - pinned | 1 - mapped)
    int memType = 0;

    // Add random material properties - copy to constant memory
    real lambda = 55.27e9;
    real mu = 25.95e9;

    std::cout << "\n---------- Constitutive update ----------" << std::endl
              << "Material properties: " << std::endl
              << "Lambda value:     " << lambda << std::endl
              << "Mu value:         " << mu << "\n"
              << std::endl;

    // Determine the number of elements and quadrature points per element
    int elem = atoi(argv[1]) * 1e6;
    int quads = 1;
    int nEval = elem * quads;
    int nAlloc = nEval * 9;

    // Log how many iterations are performed
    std::cout << "\n---------- Launch Configuration ----------" << std::endl
              << "Number of quadrature points:      " << quads << std::endl
              << "Number of elements :              " << elem / 1e6 << " M " << std::endl
              << "Number of constitutive updates:   " << nEval / 1e6 << " M" << std::endl;

    // Allocate threads for each block
    int nThreadsPerBlock = 512;

    // Set the gridsize (total number of threads) optimal amount varies per problem
    int gridsize = 400000;

    int nBlocks, nLoops;
    int nThreadsLastLoop, nBlocksLastLoop;
    if (nEval > nThreadsPerBlock)
    {
        // Retrieve the properties of the GPU
        hipDeviceProp_t prop;
        status = hipGetDeviceProperties(&prop, 0);
        checkError(status, "hipGetDeviceProperties", __FILE__, __LINE__);

        // Determine the instruction size (= amount of threads executed in parallel by all SM's)
        // There are 32 threads per SM
        int instructionSize = prop.multiProcessorCount * 32;

        // Determine the number of blocks
        nBlocks = gridsize / nThreadsPerBlock + 1;

        // Check if the last block is being used
        int nThreadsLastBlock = gridsize - (nBlocks - 1) * nThreadsPerBlock;
        if (nThreadsLastBlock == 0)
        {
            nBlocks -= 1;
            nThreadsLastBlock = nThreadsPerBlock;
        }

        // Determine the amount of loops over the saturation size to complete the problem size
        nLoops = nEval / gridsize + 1;

        // Determine the amount of blocks in the last loop
        nThreadsLastLoop = nEval - (nLoops - 1) * gridsize;

        // Check if the last loop is necessary
        int nThreadsLastLoopBlock;
        if (nThreadsLastLoop == 0)
        {
            nLoops -= 1;
            nBlocksLastLoop = nBlocks;
            nThreadsLastLoopBlock = nThreadsPerBlock;
        }
        else
        {
            // Check if the last block in the last loop is being used
            nBlocksLastLoop = nThreadsLastLoop / nThreadsPerBlock + 1;
            nThreadsLastLoopBlock = nThreadsLastLoop - (nBlocksLastLoop - 1) * nThreadsPerBlock;
            if (nThreadsLastLoopBlock == 0)
            {
                nBlocksLastLoop -= 1;
                nThreadsLastLoopBlock = nThreadsPerBlock;
            }
        }

        // Show a summary of the grid and its allocated threads
        std::cout << "\nInstruction size:                 " << instructionSize << std::endl
                  << "Used saturation amount:           " << gridsize << std::endl
                  << "Number of rounds:                 " << nLoops << std::endl
                  << "Number of blocks:                 " << nBlocks << std::endl
                  << "Number of threads:                " << nBlocks * nThreadsPerBlock << std::endl
                  << "Last block:                       " << nThreadsLastBlock << std::endl
                  << "Active blocks in last round:      " << nBlocksLastLoop << std::endl
                  << "Active threads in last round:     " << nBlocksLastLoop * nThreadsPerBlock << std::endl
                  << "Active threads in last block:     " << nThreadsLastLoopBlock << std::endl;
    }
    else
    {
        nThreadsPerBlock = nEval;
        nBlocks = 1;
        nLoops = 1;
    }

    // Create cuda timers
    hipEvent_t startComp, stopComp;
    status = hipEventCreate(&startComp);
    checkError(status, "hipEventCreate", __FILE__, __LINE__);
    status = hipEventCreate(&stopComp);
    checkError(status, "hipEventCreate", __FILE__, __LINE__);

    // Initialize variables to be stored in the GPU
    real *F, *P, *Fg, *Pg;

    if (memType == 0)
    {
        // Allocate CPU memory
        std::cout << "Running pinned implementation" << std::endl;
        status = hipHostMalloc(&F, nEval * 9 * sizeof(real));
        checkError(status, "hipHostMalloc", __FILE__, __LINE__);
        status = hipHostMalloc(&P, nEval * 9 * sizeof(real));
        checkError(status, "hipHostMalloc", __FILE__, __LINE__);

        // Allocate GPU memory
        status = hipMalloc(&Fg, nEval * 9 * sizeof(real));
        checkError(status, "hipMalloc", __FILE__, __LINE__);
        status = hipMalloc(&Pg, nEval * 9 * sizeof(real));
        checkError(status, "hipMalloc", __FILE__, __LINE__);
    }
    else
    {
        // Allocate CPU memory
        std::cout << "Running mapped implementation" << std::endl;
        status = hipHostAlloc(&F, nEval * 9 * sizeof(real), hipHostMallocMapped);
        checkError(status, "hipHostMalloc", __FILE__, __LINE__);
        status = hipHostAlloc(&P, nEval * 9 * sizeof(real), hipHostMallocMapped);
        checkError(status, "hipHostMalloc", __FILE__, __LINE__);

        // Get the pointers to the mapped memory
        status = hipHostGetDevicePointer(&Fg, F, 0);
        checkError(status, "hipHostGetDevicePointer", __FILE__, __LINE__);
        status = hipHostGetDevicePointer(&Pg, P, 0);
        checkError(status, "hipHostGetDevicePointer", __FILE__, __LINE__);
    }

    // Define an an array of random deformation gradients as a perturbation of the identity
    real F_pert = 0.1;
    getCoalescedGradients(F_pert, nEval, F, print);

    // Create cuda streams
    hipStream_t *streams;
    hipMalloc(&streams, nLoops * sizeof(hipStream_t));
    for (int i = 0; i < nLoops; i++)
    {
        status = hipStreamCreate(&(streams[i]));
        checkError(status, "hipStreamCreate", __FILE__, __LINE__);
    }

    std::cout << "\n---------- Cuda Event Timing ----------" << std::endl;

    // Executing computation
    status = hipEventRecord(startComp);
    checkError(status, "hipEventRecord", __FILE__, __LINE__);
    std::cout << "\nSynchronizing device..." << std::endl;

    // Compute the diagonal components
    int loopSize = gridsize;
    int loopBlocks = nBlocks;
    for (int i = 0; i < nLoops; i++)
    {
        // Set the size for the last loop
        if (i == (nLoops - 1))
        {
            loopSize = nThreadsLastLoop;
            loopBlocks = nBlocksLastLoop;
        }

        if (memType == 0)
        {
            // Send input data for the saturated computation
            status = hipMemcpyAsync(Fg + i * gridsize, F + i * gridsize, loopSize * sizeof(real), hipMemcpyHostToDevice, streams[i]);
            checkError(status, "hipMemcpyAsync", __FILE__, __LINE__);

            status = hipMemcpyAsync(Fg + i * gridsize + 4 * nEval, F + i * gridsize + 4 * nEval, loopSize * sizeof(real), hipMemcpyHostToDevice, streams[i]);
            checkError(status, "hipMemcpyAsync", __FILE__, __LINE__);

            status = hipMemcpyAsync(Fg + i * gridsize + 8 * nEval, F + i * gridsize + 8 * nEval, loopSize * sizeof(real), hipMemcpyHostToDevice, streams[i]);
            checkError(status, "hipMemcpyAsync", __FILE__, __LINE__);
        }

        // Perform the saturated computation
        axial<<<loopBlocks, nThreadsPerBlock, 0, streams[i]>>>(Fg + i * gridsize, lambda, mu, Pg + i * gridsize, nEval, loopSize);

        if (memType == 0)
        {
            // Receive results from the saturated computation
            status = hipMemcpyAsync(P + i * gridsize, Pg + i * gridsize, loopSize * sizeof(real), hipMemcpyDeviceToHost, streams[i]);
            checkError(status, "hipMemcpyAsync", __FILE__, __LINE__);

            status = hipMemcpyAsync(P + i * gridsize + 4 * nEval, Pg + i * gridsize + 4 * nEval, loopSize * sizeof(real), hipMemcpyDeviceToHost, streams[i]);
            checkError(status, "hipMemcpyAsync", __FILE__, __LINE__);

            status = hipMemcpyAsync(P + i * gridsize + 8 * nEval, Pg + i * gridsize + 8 * nEval, loopSize * sizeof(real), hipMemcpyDeviceToHost, streams[i]);
            checkError(status, "hipMemcpyAsync", __FILE__, __LINE__);
        }
    }

    // Compute the shear components
    int iShear[3] = {1, 2, 5};
    int jShear[3] = {3, 6, 7};
    int iComponent, jComponent;
    for (int shearComp = 0; shearComp < 3; shearComp++)
    {
        // Set the indices of the current shear components
        iComponent = iShear[shearComp];
        jComponent = jShear[shearComp];
        for (int i = 0; i < nLoops; i++)
        {
            // Set the size for the last loop
            if (i == (nLoops - 1))
            {
                loopSize = nThreadsLastLoop;
                loopBlocks = nBlocksLastLoop;
            }
            else
            {
                loopSize = gridsize;
                loopBlocks = nBlocks;
            }

            if (memType == 0)
            {
                // Send input data for the saturated computation
                status = hipMemcpyAsync(Fg + i * gridsize + iComponent * nEval, F + i * gridsize + iComponent * nEval, loopSize * sizeof(real), hipMemcpyHostToDevice, streams[i]);
                checkError(status, "hipMemcpyAsync", __FILE__, __LINE__);

                status = hipMemcpyAsync(Fg + i * gridsize + jComponent * nEval, F + i * gridsize + jComponent * nEval, loopSize * sizeof(real), hipMemcpyHostToDevice, streams[i]);
                checkError(status, "hipMemcpyAsync", __FILE__, __LINE__);
            }

            // Perform the saturated computation
            shear<<<loopBlocks, nThreadsPerBlock, 0, streams[i]>>>(Fg + i * gridsize, lambda, mu, Pg + i * gridsize, nEval, iComponent, jComponent, loopSize);

            if (memType == 0)
            {
                // Receive results from the saturated computation
                status = hipMemcpyAsync(P + i * gridsize + iComponent * nEval, Pg + i * gridsize + iComponent * nEval, loopSize * sizeof(real), hipMemcpyDeviceToHost, streams[i]);
                checkError(status, "hipMemcpyAsync", __FILE__, __LINE__);

                status = hipMemcpyAsync(P + i * gridsize + jComponent * nEval, Pg + i * gridsize + jComponent * nEval, loopSize * sizeof(real), hipMemcpyDeviceToHost, streams[i]);
                checkError(status, "hipMemcpyAsync", __FILE__, __LINE__);
            }
        }
    }

    // Wait for GPU to finish before accessing on host
    status = hipDeviceSynchronize();
    checkError(status, "hipDeviceSynchronize", __FILE__, __LINE__);

    // Check timing of executing computation
    status = hipEventRecord(stopComp);
    checkError(status, "hipEventRecord", __FILE__, __LINE__);
    status = hipEventSynchronize(stopComp);
    checkError(status, "hipEventSynchronize", __FILE__, __LINE__);

    // Compute the elapsed time
    float milliseconds = 0.;
    status = hipEventElapsedTime(&milliseconds, startComp, stopComp);
    checkError(status, "hipEventElapsedTime", __FILE__, __LINE__);

    std::cout << "Finished synchronizing device" << std::endl
              << "\nExecuting computation took: " << milliseconds << " ms" << std::endl;

    std::cout << "\n---------- Cuda Stop Timing ----------" << std::endl;

    // Destroy events
    status = hipEventDestroy(startComp);
    checkError(status, "hipEventDestroy", __FILE__, __LINE__);
    status = hipEventDestroy(stopComp);
    checkError(status, "hipEventDestroy", __FILE__, __LINE__);

    // Destroy the cuda streams
    for (int i = 0; i < nLoops; i++)
    {
        status = hipStreamDestroy(streams[i]);
        checkError(status, "hipStreamDestroy", __FILE__, __LINE__);
    }

    // Verify each result is at the correct index
    std::cout << "\nStarting verification by index..." << std::endl;
    verifyCoalescedResults(nEval, lambda, mu, F, P, print);
    std::cout << "Finished verification by index..." << std::endl;

    // In case of pinned memory also free the GPU memory
    if (memType == 0)
    {
        status = hipFree(Fg);
        checkError(status, "hipFree", __FILE__, __LINE__);

        status = hipFree(Pg);
        checkError(status, "hipFree", __FILE__, __LINE__);
    }

    status = hipHostFree(F);
    checkError(status, "hipHostFree", __FILE__, __LINE__);
    status = hipHostFree(P);
    checkError(status, "hipHostFree", __FILE__, __LINE__);

    // Reset the device in the end
    status = hipDeviceReset();
    checkError(status, "hipDeviceReset", __FILE__, __LINE__);

    return 0;
}